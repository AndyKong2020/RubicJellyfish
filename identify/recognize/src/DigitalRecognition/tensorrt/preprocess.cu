#include "hip/hip_runtime.h"
#include "preprocess.h"

__global__ void preprocess_kernel(INPUT_VAR_TYPE* input, float* output, int size) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= size) return;
    output[position] = ((float)input[position] - MEAN) / SCALE;
}

void cuda_preprocess(INPUT_VAR_TYPE* input, float* output, int size, hipStream_t stream) {
    int threads = 256;
    int blocks = size / (float)threads;
    preprocess_kernel<<<blocks, threads, 0, stream>>>(input, output, size);
}
